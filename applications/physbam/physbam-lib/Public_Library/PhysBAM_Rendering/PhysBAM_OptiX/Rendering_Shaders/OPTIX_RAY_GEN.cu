#include "hip/hip_runtime.h"
//#####################################################################
// Copyright 2011, Valeria Nikolaenko, Rahul Sheth
// This file is part of PhysBAM whose distribution is governed by the license contained in the accompanying file PHYSBAM_COPYRIGHT.txt.
//#####################################################################
#include <optix_world.h>
#include "OPTIX_HELPERS.h"
#include "OPTIX_RAY_STRUCTS.h"
using namespace optix;

rtDeclareVariable(float2,halfed_proj_metrics,,);
rtDeclareVariable(uint2,screen_metrics,,);
rtDeclareVariable(float3,up,,);
rtDeclareVariable(float3,right,,);
rtDeclareVariable(float3,dir,,);
rtDeclareVariable(float3,loc,,);
rtDeclareVariable(float3,from_loc_to_focal,,);
rtDeclareVariable(uint2,launch_index,rtLaunchIndex,);
rtDeclareVariable(uint2,launch_dim,rtLaunchDim,);
rtDeclareVariable(rtObject,top_object,,);
rtDeclareVariable(float,scene_epsilon,,);
rtBuffer<uchar4, 2> output_buffer;

RT_PROGRAM void ray_gen() 
{
    float2 d=make_float2(launch_index)/make_float2(launch_dim)*2.f-1.f;
    float3 ray_origin=loc;
    float3 ray_direction=normalize(d.x*right*halfed_proj_metrics.x+d.y*up*halfed_proj_metrics.y+from_loc_to_focal);
    optix::Ray ray=optix::make_Ray(ray_origin,ray_direction,0,scene_epsilon,RT_DEFAULT_MAX);

    PerRayData_radiance prd;
    prd.depth=0;
    prd.importance=1;

    rtTrace(top_object,ray,prd);
    output_buffer[launch_index]=make_color(prd.result);
}

RT_PROGRAM void exception() 
{
  const unsigned int code=rtGetExceptionCode();
  rtPrintExceptionDetails();
  rtPrintf("Caught exception 0x%X at launch index (%d,%d)\n",code,launch_index.x,launch_index.y);
  output_buffer[launch_index]=make_uchar4(1.0f,1.0f,1.0f,0);
}

