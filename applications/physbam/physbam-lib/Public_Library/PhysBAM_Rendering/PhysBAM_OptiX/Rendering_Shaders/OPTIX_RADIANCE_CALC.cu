#include "hip/hip_runtime.h"
//#####################################################################
// Copyright 2011, Valeria Nikolaenko, Rahul Sheth, Bo Zhu.
// This file is part of PhysBAM whose distribution is governed by the license contained in the accompanying file PHYSBAM_COPYRIGHT.txt.
//#####################################################################
// OptiX phong material program radiance_calc.cu
//#####################################################################

#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>
#include <optix.h>
#include <optixu/optixpp_namespace.h>
#include <optix_world.h>
#include "OPTIX_COMMONSTRUCTS.h"
#include "OPTIX_HELPERS.h"
#include "OPTIX_RAY_STRUCTS.h"

using namespace optix;

rtDeclareVariable(float3,Ka,,);
rtDeclareVariable(float3,Kd,,);
rtDeclareVariable(float3,Ks,,);
rtDeclareVariable(float3,Kt,,);
rtDeclareVariable(float3,reflectivity,,);
rtDeclareVariable(float,phong_exp,,);
rtDeclareVariable(float,refract_coef,,);
rtDeclareVariable(float,transpar_atten_coef,,);
rtDeclareVariable(float3,cutoff_color,,);
rtDeclareVariable(float3,ambient_light_color,,);

rtDeclareVariable(float3,shading_normal,attribute shading_normal,);
rtDeclareVariable(float3,geometric_normal,attribute geometric_normal,);
rtDeclareVariable(int,max_depth,,);
rtDeclareVariable(float,scene_epsilon,,);

rtDeclareVariable(float,t_hit,rtIntersectionDistance,);
rtDeclareVariable(float3,extinction_constant,,);
rtDeclareVariable(rtObject,top_object,,);
rtDeclareVariable(rtObject,top_opaque_object,,);

rtDeclareVariable(optix::Ray,ray,rtCurrentRay,);
rtDeclareVariable(PerRayData_radiance,prd,rtPayload,);
rtDeclareVariable(PerRayData_shadow,prd_shadow,rtPayload,);

rtBuffer<BasicLight> lights;

RT_PROGRAM void any_hit_shadow_phong() 
{
    rtIgnoreIntersection();
    prd_shadow.attenuation*=0.5f;
    rtTerminateRay();
}

RT_PROGRAM void closest_hit_radiance_phong() {
  float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

  prd.distance = t_hit;
  float3 hit_point = ray.origin + t_hit * ray.direction;
  float3 beer_attenuation = make_float3(1);

  float3 result = make_float3(0.f);
  result += Ka * ambient_light_color;

  // compute direct lighting
  uint num_lights = lights.size();
  for (int i = 0; i < num_lights; ++i) {
    BasicLight light = lights[i];
    float Ldist = optix::length(light.pos - hit_point);
    float3 L = optix::normalize(light.pos - hit_point);
    float nDl = optix::dot(world_shading_normal, L);

    // cast shadow ray
    float3 light_attenuation = make_float3(static_cast<float>( nDl > 0.0f ));
    /*
    // no shadows for speeding
    if (nDl > 0.0f && light.casts_shadow) {
      PerRayData_shadow shadow_prd;
      shadow_prd.attenuation = make_float3(1.0f);
      optix::Ray shadow_ray = optix::make_Ray(fhp, L, 1, scene_epsilon, Ldist);
      rtTrace(top_opaque_object, shadow_ray, shadow_prd);
      light_attenuation = shadow_prd.attenuation;
    }
    */

    // If not completely shadowed, light the hit point
    if (fmaxf(light_attenuation) > 0.0f) {
      float3 Lc = light.color * light_attenuation;
      result += Kd * nDl * Lc;

      float3 H = optix::normalize(L - ray.direction);
      float nDh = optix::dot(world_shading_normal, H);
      if(nDh > 0 && fmaxf(Ks) > 0) {
        float power = pow(nDh, phong_exp);
        result += Ks * power * Lc;
      }
    }
  }

  if (fmaxf(reflectivity) > 0 && prd.depth < max_depth) {
    // ray tree attenuation
    PerRayData_radiance new_prd;
    new_prd.importance = prd.importance * optix::luminance(reflectivity);
    new_prd.depth = prd.depth + 1;
    new_prd.result = make_float3(0.f, 0.f, 0.f);

    // reflection ray
    if (new_prd.importance > 0.01f) {
      float3 R = optix::reflect(ray.direction, world_shading_normal);
      optix::Ray refl_ray = optix::make_Ray(hit_point, R, 0, scene_epsilon, RT_DEFAULT_MAX);
      rtTrace(top_object, refl_ray, new_prd);
      result += reflectivity * new_prd.result * beer_attenuation;
    }
  }

  // Refraction
  if (fmaxf(Kt) > 0.0f && prd.depth < max_depth) {

    float3 t;
    if (refract(t, ray.direction, world_shading_normal, refract_coef)) {
      // check for external or internal reflection
      float cos_theta = dot(ray.direction, world_shading_normal);
      if (cos_theta < 0.0f)
        cos_theta = -cos_theta;
      else
        cos_theta = dot(t, world_shading_normal);

      // float fresnel_exponent = 3.0f, fresnel_minimum = 1.0f, fresnel_maximum = 1.0f

      float reflection = 0.f;//fresnel_schlick(cos_theta, 4.0f, 0.1f, 1.f);
      PerRayData_radiance new_prd;
      new_prd.importance = prd.importance * (1.0f-reflection) * optix::luminance(Kt * beer_attenuation);
      new_prd.depth = prd.depth + 1;

      if (new_prd.importance > 0.01f) {
        optix::Ray refl_ray = optix::make_Ray(hit_point, t, 0, scene_epsilon, RT_DEFAULT_MAX);
        rtTrace(top_object, refl_ray, new_prd);
        // attenuate by depth if we are coming into the surface
        if (new_prd.distance >= 0 && dot(world_shading_normal, ray.direction) < 0) {
            result += (1.0f-reflection) * Kt * beer_attenuation * new_prd.result * lerp(cutoff_color/*make_float3(0.3f, 0.7f, 0.9f)*/, make_float3(1.0f, 1.0f, 1.0f), exp(-transpar_atten_coef * new_prd.distance));
        } else {
            result += (1.0f-reflection) * Kt * beer_attenuation * new_prd.result;
        }
      }
    } else { // full inner reflection
        // ray tree attenuation
        PerRayData_radiance new_prd;
        new_prd.importance = prd.importance * optix::luminance(Kt);
        new_prd.depth = prd.depth + 1;
        new_prd.result = make_float3(0.f, 0.f, 0.f);

        // reflection ray
        if (new_prd.importance > 0.01f) {
          float3 R = optix::reflect(ray.direction, world_shading_normal);
          optix::Ray refl_ray = optix::make_Ray(hit_point, R, 0, scene_epsilon, RT_DEFAULT_MAX);
          rtTrace(top_object, refl_ray, new_prd);
          if (new_prd.distance >= 0 && dot(world_shading_normal, ray.direction) < 0) {
            result += Kt * new_prd.result * beer_attenuation * lerp(cutoff_color, make_float3(1.0f, 1.0f, 1.0f), exp(-transpar_atten_coef * new_prd.distance));
          } else {
            result += Kt * new_prd.result * beer_attenuation;
          }
        }
    }
  }

  // cutoff color
  if (prd.depth >= max_depth && fmaxf(result) == 0) {
      result = cutoff_color;
  }
  prd.result = result;
}

RT_PROGRAM void any_hit_shadow_lambertian()
{
  prd_shadow.attenuation=make_float3(0);
  rtTerminateRay();
}

RT_PROGRAM void closest_hit_radiance_floor_lambertian()
{
    float3 ffnormal=make_float3(0,1.0f,0);
    float3 color=Ka*ambient_light_color;

    float3 hit_point=ray.origin+t_hit*ray.direction;
    for(int i=0;i<lights.size();i++){
        BasicLight light=lights[i];
        float3 light_hit_dir=normalize(light.pos-hit_point);
        float lam=dot(ffnormal,light_hit_dir);
        if(lam>0)color+=Kd*lam*light.color;
    }
    prd.result=color;
}

RT_PROGRAM void closest_hit_radiance_floor_lambertian_with_shadow()
{
    float3 ffnormal=make_float3(0,1.0f,0);
    float3 color=Ka*ambient_light_color;

    float3 hit_point=ray.origin+t_hit*ray.direction;
    for(int i=0;i<lights.size();i++){
        BasicLight light=lights[i];
        float3 light_hit_dir=normalize(light.pos-hit_point);
        float light_hit_distance=length(light.pos-hit_point);
        float lam=dot(ffnormal,light_hit_dir);
        if(lam>0){
            optix::Ray ray_shadow=optix::make_Ray(light.pos,-light_hit_dir,RAY_TYPE_SHADOW,RAY_T_MIN,light_hit_distance-RAY_T_MIN);
			PerRayData_shadow prd_shadow;prd_shadow.attenuation=make_float3(1.0f);
			rtTrace(top_object,ray_shadow,prd_shadow);
			float3 light_attenuation=prd_shadow.attenuation;
			color+=Kd*lam*light.color*light_attenuation;
        }
    }
    prd.result=color;
}

RT_PROGRAM void closest_hit_radiance_lambertian()
{
    float3 world_shading_normal=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,shading_normal));
    float3 world_geometric_normal=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,geometric_normal));
    float3 ffnormal=faceforward(world_shading_normal,-ray.direction,world_geometric_normal);
    float3 color=Ka*ambient_light_color;

    float3 hit_point=ray.origin+t_hit*ray.direction;
    for(int i=0;i<lights.size();i++){
        BasicLight light=lights[i];
        float3 light_hit_dir=normalize(light.pos-hit_point);
        float lam=dot(ffnormal,light_hit_dir);
        if(lam>0)color+=Kd*lam*light.color;
    }
    prd.result=color;
}

RT_PROGRAM void closest_hit_radiance_lambertian_with_shadow()
{
    float3 world_shading_normal=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,shading_normal));
    float3 world_geometric_normal=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,geometric_normal));
    float3 ffnormal=faceforward(world_shading_normal,-ray.direction,world_geometric_normal);
    float3 color=Ka*ambient_light_color;

    float3 hit_point=ray.origin+t_hit*ray.direction;
    for(int i=0;i<lights.size();i++){
        BasicLight light=lights[i];
        float3 light_hit_dir=normalize(light.pos-hit_point);
        float light_hit_distance=length(light.pos-hit_point);
        float lam=dot(ffnormal,light_hit_dir);
        if(lam>0){
            optix::Ray ray_shadow=optix::make_Ray(light.pos,-light_hit_dir,RAY_TYPE_SHADOW,RAY_T_MIN,light_hit_distance-RAY_T_MIN);
			PerRayData_shadow prd_shadow;prd_shadow.attenuation=make_float3(1.0f);
			rtTrace(top_object,ray_shadow,prd_shadow);
			float3 light_attenuation=prd_shadow.attenuation;
			color+=Kd*lam*light.color*light_attenuation;
        }
    }
    prd.result=color;
}