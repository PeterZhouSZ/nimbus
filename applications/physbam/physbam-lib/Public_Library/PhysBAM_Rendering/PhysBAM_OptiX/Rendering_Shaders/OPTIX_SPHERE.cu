#include "hip/hip_runtime.h"
//#####################################################################
// Copyright 2011, Valeria Nikolaenko, Rahul Sheth
// This file is part of PhysBAM whose distribution is governed by the license contained in the accompanying file PHYSBAM_COPYRIGHT.txt.
//#####################################################################
#include <optix_world.h>
#include "OPTIX_HELPERS.h"
using namespace optix;

rtDeclareVariable(float3,center,,);
rtDeclareVariable(float,radius,,);
rtDeclareVariable(float3,geometric_normal,attribute geometric_normal,);
rtDeclareVariable(float3,shading_normal,attribute shading_normal,);
rtDeclareVariable(float,scene_epsilon,,);
rtDeclareVariable(optix::Ray,ray,rtCurrentRay,);

RT_PROGRAM void intersect(int) 
{
  float3 O=ray.origin-center;
  float3 D=ray.direction;

  float b=dot(O, D);
  float c=dot(O, O)-radius*radius;
  float disc=b*b-c;
  if(disc>0.0f){
    float sdisc=sqrtf(disc);
    float root1=(-b-sdisc);
    bool check_second=true;
    if(rtPotentialIntersection(root1)){
      shading_normal=geometric_normal=(O + root1*D)/radius;
      float3 hit_p=ray.origin+root1*ray.direction;
      float3 offset=shading_normal*scene_epsilon;
      if(rtReportIntersection(0))check_second = false;
    }
    if(check_second){
      float root2=(-b+sdisc);
      if(rtPotentialIntersection(root2)){
        shading_normal=geometric_normal=(O+root2*D)/radius;
        float3 hit_p=ray.origin+root2*ray.direction;
        float3 offset=shading_normal*scene_epsilon;
        float t=dot(shading_normal,ray.direction)<0?1:-1;
        rtReportIntersection(0);
      }
    }
  }
}

RT_PROGRAM void box_bounds(int,float result[6]) 
{
  const float3 rad=make_float3(radius);
  optix::Aabb* aabb=(optix::Aabb*)result;

  if(radius>0.0f&&!isinf(radius)){
    aabb->m_min=center-rad;
    aabb->m_max=center+rad;
  } 
  else{
    aabb->invalidate();
  }
}
