#include "hip/hip_runtime.h"
//#####################################################################
// Copyright 2011, Valeria Nikolaenko, Rahul Sheth
// This file is part of PhysBAM whose distribution is governed by the license contained in the accompanying file PHYSBAM_COPYRIGHT.txt.
//#####################################################################

#include <optix_world.h>
#include "OPTIX_HELPERS.h"
#include "OPTIX_INTERSECTION_REFINEMENT.h"
#include "OPTIX_RAY_STRUCTS.h"
using namespace optix;

rtBuffer<float3> vertex_buffer;
rtBuffer<uint3> vindex_buffer;

rtBuffer<float3> normal_buffer;
// rtBuffer<uint3> nindex_buffer;

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, front_hit_point, attribute front_hit_point, );
rtDeclareVariable(float3, back_hit_point, attribute back_hit_point, );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(int, interpolate_normal, , );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );

RT_PROGRAM void intersect(int primIdx) {
  uint3 v_idx = vindex_buffer[primIdx];

  float3 p0 = vertex_buffer[ v_idx.x ];
  float3 p1 = vertex_buffer[ v_idx.y ];
  float3 p2 = vertex_buffer[ v_idx.z ];

  // Intersect ray with triangle
  float3 n;
  float  t, beta, gamma;
  if( intersect_triangle( ray, p0, p1, p2, n, t, beta, gamma ) ) {

    if(  rtPotentialIntersection( t ) ) {

      uint3 n_idx = v_idx;//nindex_buffer[ primIdx ];

      if ( normal_buffer.size() == 0 || n_idx.x < 0 || n_idx.y < 0 || n_idx.z < 0 ) {
        shading_normal = normalize( n );
      } else {
        float3 n0 = normal_buffer[ n_idx.x ];
        float3 n1 = normal_buffer[ n_idx.y ];
        float3 n2 = normal_buffer[ n_idx.z ];
        if (interpolate_normal) {
            shading_normal = normalize( n1*beta + n2*gamma + n0*(1.0f-beta-gamma) );
        } else {
            shading_normal = normalize( n );
        }
      }
      /* shading_normal = */geometric_normal = normalize( n );

      back_hit_point = front_hit_point = ray.origin + t * ray.direction;
      /*
      if (dot(ray.direction, shading_normal) > 0.f) {
        back_hit_point = front_hit_point - shading_normal * scene_epsilon;
        front_hit_point = front_hit_point + shading_normal * scene_epsilon;
      } else {
        back_hit_point = front_hit_point + shading_normal * scene_epsilon;
        front_hit_point = front_hit_point - shading_normal * scene_epsilon;
      }*/

      // refine_and_offset_hitpoint( ray.origin + t*ray.direction, ray.direction, geometric_normal, p0, back_hit_point, front_hit_point );
      // back_hit_point = front_hit_point = ray.origin + t * ray.direction;// + shading_normal * scene_epsilon;

      /*
      geometric_normal = shading_normal = normalize(n);
      int3 t_idx = tindex_buffer[ primIdx ];
      if ( texcoord_buffer.size() == 0 || t_idx.x < 0 || t_idx.y < 0 || t_idx.z < 0 ) {
        texcoord = make_float3( 0.0f, 0.0f, 0.0f );
      } else {
        float2 t0 = texcoord_buffer[ t_idx.x ];
        float2 t1 = texcoord_buffer[ t_idx.y ];
        float2 t2 = texcoord_buffer[ t_idx.z ];
        texcoord = make_float3( t1*beta + t2*gamma + t0*(1.0f-beta-gamma) );
      }

      rtReportIntersection(material_buffer[primIdx]);
      */
      rtReportIntersection(0);
    }
  }
}

RT_PROGRAM void box_bounds (int primIdx, float result[6]) {
  const uint3 v_idx = vindex_buffer[primIdx];

  const float3 v0   = vertex_buffer[ v_idx.x ];
  const float3 v1   = vertex_buffer[ v_idx.y ];
  const float3 v2   = vertex_buffer[ v_idx.z ];

  const float  area = length(cross(v1-v0, v2-v0));
  optix::Aabb* aabb = (optix::Aabb*)result;

  if(area > 0.0f && !isinf(area)) {
    aabb->m_min = fminf( fminf( v0, v1), v2 );
    aabb->m_max = fmaxf( fmaxf( v0, v1), v2 );
  } else {
    aabb->invalidate();
  }
}

