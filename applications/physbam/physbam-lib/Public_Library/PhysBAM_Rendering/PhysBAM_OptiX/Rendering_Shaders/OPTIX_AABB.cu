#include "hip/hip_runtime.h"
//#####################################################################
// Copyright 2011, Valeria Nikolaenko, Rahul Sheth
// This file is part of PhysBAM whose distribution is governed by the license contained in the accompanying file PHYSBAM_COPYRIGHT.txt.
//#####################################################################

#include <optix_world.h>
#include "OPTIX_HELPERS.h"
using namespace optix;

rtDeclareVariable(float3,low_corner,,);
rtDeclareVariable(float3,up_corner,,);
rtDeclareVariable(optix::Ray,ray,rtCurrentRay,);
rtDeclareVariable(float3,geometric_normal,attribute geometric_normal,);
rtDeclareVariable(float3,shading_normal,attribute shading_normal,);

__device__ float3 boxnormal(float t) 
{
  float3 t0=(low_corner-ray.origin)/ray.direction;
  float3 t1=(up_corner-ray.origin)/ray.direction;
  float3 neg=make_float3(t==t0.x?1:0,t==t0.y?1:0,t==t0.z?1:0);
  float3 pos=make_float3(t==t1.x?1:0,t==t1.y?1:0,t==t1.z?1:0);
  return pos-neg;
}

RT_PROGRAM void intersect(int) 
{
    float3 t1=(up_corner-ray.origin)/ray.direction;
    float3 t2=(low_corner-ray.origin)/ray.direction;
    float3 T1=fminf(t1,t2);
    float3 T2=fmaxf(t1,t2);
    float Tnear=fmaxf(T1);
    float Tfar=fminf(T2);

    if(Tnear<=Tfar){
        bool checkAnother=true;
        if(rtPotentialIntersection(Tnear)){
            shading_normal=geometric_normal=boxnormal(Tnear);
            if(rtReportIntersection(0))checkAnother=false;
        }
        if(checkAnother&&rtPotentialIntersection(Tfar)){
            shading_normal=geometric_normal=boxnormal(Tfar);
            rtReportIntersection(0);
        }
    }
}

RT_PROGRAM void box_bounds(int,float result[6]) 
{
  optix::Aabb* aabb=(optix::Aabb*)result;
  aabb->m_min=low_corner;
  aabb->m_max=up_corner;
}

