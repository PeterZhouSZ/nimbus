#include "hip/hip_runtime.h"
//#####################################################################
// Copyright 2011, Valeria Nikolaenko
// This file is part of PhysBAM whose distribution is governed by the license contained in the accompanying file PHYSBAM_COPYRIGHT.txt.
//#####################################################################

#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>
#include "OPTIX_COMMONSTRUCTS.h"
#include "OPTIX_HELPERS.h"
#include "OPTIX_RAY_STRUCTS.h"
using namespace optix;

rtTextureSampler<float, 3, hipReadModeElementType> smoke_dencities_tex;

rtDeclareVariable(float3,       low_corner, , );
rtDeclareVariable(float3,       up_corner, , );
rtDeclareVariable(float,       scene_epsilon, , );
rtDeclareVariable(float,       step, , );
rtDeclareVariable(float,       color_multiplier, , );
// rtDeclareVariable(float,       exp_multiplier, , );

// rtDeclareVariable(rtObject,          smoke_object, , );
rtDeclareVariable(rtObject,          top_opaque_object, , );

rtDeclareVariable(PerRayData_photon, prd, rtPayload, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

//rtDeclareVariable(float, distance, attribute distance, );
rtDeclareVariable(float, front_box_hit_point, attribute front_box_hit_point, );

__device__ bool isInsideDomain(float3 point) {
    return !(point.x < low_corner.x || point.x > up_corner.x ||
        point.y < low_corner.y || point.y > up_corner.y ||
        point.z < low_corner.z || point.z > up_corner.z);
}

__device__ float3 operator*(float3 a, uint3 b) {
    return make_float3(a.x * b.x, a.y * b.y, a.z * b.z);
}

__device__ float getIntensity(float3 point) {
    if (!isInsideDomain(point)) {
        return -1.f;
    }
    float3 tex_coord = (point - low_corner) / (up_corner - low_corner);
    return tex3D(smoke_dencities_tex, tex_coord.x, tex_coord.y, tex_coord.z);
}

RT_PROGRAM void any_hit_shadow() {
  // phongShadowed();
  rtIgnoreIntersection();
}

RT_PROGRAM void closest_hit_radiance() {
  float3 hit_point = ray.origin + (t_hit + scene_epsilon) * ray.direction;

  /*
  float t = t_hit - front_box_hit_point;
  if (t > scene_epsilon || t < -scene_epsilon) {
    prd.result = make_float3(1.f, 0.f, 0.f);
    return;
  }*/

  PerRayData_radiance new_prd;
  new_prd.depth = 0;
  new_prd.importance = 1.f;

  optix::Ray next_ray = optix::make_Ray(ray.origin, ray.direction, 0, scene_epsilon, RT_DEFAULT_MAX);
  rtTrace(top_opaque_object, next_ray, new_prd);

  if (!isInsideDomain(hit_point)) {
     prd.result = 0.f;
     return;
  }

  if (new_prd.distance > 0 && new_prd.distance < prd.stop_distance) {
    prd.result = 0.f;
    return;
  }

  float density = getIntensity(hit_point);
  float accumulative_density = 0;
  float accumulative_step = front_box_hit_point;
  do {
     accumulative_density += density * step;
     // hit_point += ray.direction * step;
     density = getIntensity(hit_point);
     accumulative_step += step;
  } while (density >= 0.f && accumulative_step < prd.stop_distance);

  float res = color_multiplier * accumulative_density;//exp(-density_multiplier * accumulative_density);
  if (res > 1.0f) {
    res = 1.0f;
  } else if (res < 0.f) {
    res = 0.f;
  }
  prd.result = /*getIntensity(ray.origin + prd.stop_distance * ray.direction) * */(1.0f - res);
}
