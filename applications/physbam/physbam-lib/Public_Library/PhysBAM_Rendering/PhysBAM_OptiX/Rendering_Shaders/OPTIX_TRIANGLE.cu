#include "hip/hip_runtime.h"
//#####################################################################
// Copyright 2011, Valeria Nikolaenko, Rahul Sheth
// This file is part of PhysBAM whose distribution is governed by the license contained in the accompanying file PHYSBAM_COPYRIGHT.txt.
//#####################################################################

#include <optix_world.h>
#include "OPTIX_HELPERS.h"

using namespace optix;

rtDeclareVariable(float3, v1, , );
rtDeclareVariable(float3, v2, , );
rtDeclareVariable(float3, v3, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
// rtDeclareVariable(float3, back_hit_point, attribute back_hit_point, );
// rtDeclareVariable(float3, front_hit_point, attribute front_hit_point, );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

RT_PROGRAM void intersect(int) {
  // Intersect ray with triangle
  float3 n;
  float  t, beta, gamma;

  if(intersect_triangle(ray, v1, v2, v3, n, t, beta, gamma)) {
    if(rtPotentialIntersection(t)) {
      shading_normal = geometric_normal = normalize( n );

      // back_hit_point = front_hit_point = ray.origin + t * ray.direction + shading_normal * scene_epsilon;
      rtReportIntersection(0);
    }
  }
}

RT_PROGRAM void box_bounds (int, float result[6]) {
  optix::Aabb* aabb = (optix::Aabb*)result;

  aabb->m_min = fminf( fminf( v1, v2), v3 );
  aabb->m_max = fmaxf( fmaxf( v1, v2), v3 );
}
