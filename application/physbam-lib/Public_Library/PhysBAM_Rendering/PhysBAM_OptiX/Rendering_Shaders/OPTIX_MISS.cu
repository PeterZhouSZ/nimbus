#include "hip/hip_runtime.h"
//#####################################################################
// Copyright 2011, Valeria Nikolaenko, Rahul Sheth
// This file is part of PhysBAM whose distribution is governed by the license contained in the accompanying file PHYSBAM_COPYRIGHT.txt.
//#####################################################################
#include <optix_world.h>
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "OPTIX_RAY_STRUCTS.h"
using namespace optix;

rtDeclareVariable(float3,bg_color,,);

rtDeclareVariable(PerRayData_radiance,prd_radiance,rtPayload,);
rtDeclareVariable(optix::Ray,ray,rtCurrentRay,);

RT_PROGRAM void miss() 
{
    prd_radiance.result=bg_color;
}
