#include "hip/hip_runtime.h"
//#####################################################################
// Copyright 2011, Valeria Nikolaenko, Rahul Sheth
// This file is part of PhysBAM whose distribution is governed by the license contained in the accompanying file PHYSBAM_COPYRIGHT.txt.
//#####################################################################
#include <optix_world.h>
#include "OPTIX_HELPERS.h"
using namespace optix;

rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, phi_tex_delta, , );

rtTextureSampler<float, 3, hipReadModeElementType> phi_tex;

// rtDeclareVariable(float3, back_hit_point, attribute back_hit_point, );
// rtDeclareVariable(float3, front_hit_point, attribute front_hit_point, );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
__device__ bool isInsideDomain(float3 point) {
    float3 up_corner = make_float3(1.f, 1.f, 1.f);
    float3 low_corner = make_float3(0.f, 0.f, 0.f);

    return !(point.x < low_corner.x || point.x > up_corner.x ||
        point.y < low_corner.y || point.y > up_corner.y ||
        point.z < low_corner.z || point.z > up_corner.z);
}

__device__ float3 operator*(float3 a, uint3 b) {
    return make_float3(a.x * b.x, a.y * b.y, a.z * b.z);
}

__device__ float getIntensity(float3 point) {
    float3 up_corner = make_float3(1.f, 1.f, 1.f);
    float3 low_corner = make_float3(0.f, 0.f, 0.f);

    if (!isInsideDomain(point)) {
        return -1.f;
    }
    float3 tex_coord = (point - low_corner) / (up_corner - low_corner);
    // tex_coord = tex_coord * 2 - make_float3(1.f, 1.f, 1.f);
    // return tex_coord.x*tex_coord.x + tex_coord.y*tex_coord.y + tex_coord.z*tex_coord.z - 1;
    return tex3D(phi_tex, tex_coord.x, tex_coord.y, tex_coord.z);
}

__device__ float3 boxnormal(float t) {
  float3 up_corner = make_float3(1.f, 1.f, 1.f);
  float3 low_corner = make_float3(0.f, 0.f, 0.f);

  float3 t0 = (low_corner - ray.origin)/ray.direction;
  float3 t1 = (up_corner - ray.origin)/ray.direction;
  float3 neg = make_float3(t==t0.x?1:0, t==t0.y?1:0, t==t0.z?1:0);
  float3 pos = make_float3(t==t1.x?1:0, t==t1.y?1:0, t==t1.z?1:0);
  return pos-neg;
}

__device__ float intersect_box(optix::Ray ray, float &nearS, float &farS) {
    float3 up_corner = make_float3(1.f, 1.f, 1.f);
    float3 low_corner = make_float3(0.f, 0.f, 0.f);

    float3 t1 = (up_corner - ray.origin) / ray.direction;
    float3 t2 = (low_corner - ray.origin) / ray.direction;

    float3 T1 = fminf(t1, t2);
    float3 T2 = fmaxf(t1, t2);

    float Tnear = fmaxf(T1);
    float Tfar = fminf(T2);

    if (Tnear < Tfar && Tfar > 0) {
        if (Tnear < 0) {
            Tnear = scene_epsilon * 10;
        }
        /*
        shading_normal = geometric_normal = boxnormal(Tnear);
        if(rtPotentialIntersection(Tnear)) {
            shading_normal = geometric_normal = boxnormal(Tnear);
            front_hit_point = ray.origin + Tnear * ray.direction + geometric_normal * scene_epsilon;
            back_hit_point = front_hit_point - 2 * scene_epsilon * geometric_normal;
            rtReportIntersection(0);
        }
        */
        nearS = Tnear;
        farS = Tfar;
        return Tnear;
    }
    return -1;
}

RT_PROGRAM void intersect(int) {
    float d, d_far;
    intersect_box(ray, d, d_far);
    float step = fminf(phi_tex_delta);
    if (d > 0) {
        float3 hit_point = ray.origin + (d + scene_epsilon) * ray.direction;
        bool t = !isInsideDomain(ray.origin) || getIntensity(ray.origin) > 0;
        float current_distance;

        while (d < d_far) {
            current_distance = getIntensity(hit_point);
            if ((t && current_distance < 0) || (!t && current_distance > 0)) {
                if(rtPotentialIntersection(d)) {
                    shading_normal = geometric_normal = normalize(make_float3((getIntensity(hit_point + make_float3(phi_tex_delta.x, 0.f, 0.f)) - getIntensity(hit_point - make_float3(phi_tex_delta.x, 0.f, 0.f))) / (2 * phi_tex_delta.x),
                                                                              (getIntensity(hit_point + make_float3(0.f, phi_tex_delta.y, 0.f)) - getIntensity(hit_point - make_float3(0.f, phi_tex_delta.y, 0.f))) / (2 * phi_tex_delta.y),
                                                                              (getIntensity(hit_point + make_float3(0.f, 0.f, phi_tex_delta.z)) - getIntensity(hit_point - make_float3(0.f, 0.f, phi_tex_delta.z))) / (2 * phi_tex_delta.z)));
                    // shading_normal = geometric_normal = make_float3(0.f, 1.f, 0.f);
                    /*
                    front_hit_point = hit_point + scene_epsilon * geometric_normal;
                    back_hit_point = hit_point - scene_epsilon * geometric_normal;
                    */
                    // back_hit_point = front_hit_point = hit_point;
                    rtReportIntersection(0);
                }
                d -= step;
                step *= 0.1;
                if (step / 2 < 1e-4)
                    break;
            }
            d += step;
            hit_point = ray.origin + d * ray.direction;
        }
    }
    // shading_normal = geometric_normal = front_hit_point = back_hit_point = make_float3(0.f, 0.f, 0.f);
}

RT_PROGRAM void box_bounds (int, float result[6]) {
  optix::Aabb* aabb = (optix::Aabb*)result;

  aabb->m_min = make_float3(0, 0, 0);
  aabb->m_max = make_float3(1, 1, 1);
}


