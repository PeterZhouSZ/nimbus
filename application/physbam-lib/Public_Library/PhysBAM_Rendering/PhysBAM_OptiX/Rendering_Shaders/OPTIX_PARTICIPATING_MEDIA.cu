#include "hip/hip_runtime.h"
//#####################################################################
// Copyright 2012, Bo Zhu
// This file is part of PhysBAM whose distribution is governed by the license contained in the accompanying file PHYSBAM_COPYRIGHT.txt.
//#####################################################################
#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>
#include "OPTIX_COMMONSTRUCTS.h"
#include "OPTIX_HELPERS.h"
#include "OPTIX_RAY_STRUCTS.h"
using namespace optix;

rtDeclareVariable(float,front_box_hit_t,attribute front_box_hit_t,);
rtDeclareVariable(float,back_box_hit_t,attribute back_box_hit_t,);
rtDeclareVariable(optix::Ray,ray,rtCurrentRay,);
rtDeclareVariable(PerRayData_radiance,prd,rtPayload,);
rtDeclareVariable(PerRayData_shadow,prd_shadow,rtPayload,);
rtDeclareVariable(float3,low_corner,,);
rtDeclareVariable(float3,up_corner,,);
rtDeclareVariable(float3,one_over_box_size,,);
rtDeclareVariable(float,t_step,,)=0.016f;
rtDeclareVariable(float,t_step_2,,)=0.016f;
rtDeclareVariable(rtObject,top_opaque_object,,);

rtTextureSampler<float,3,hipReadModeElementType> soot_texture;
rtTextureSampler<float,3,hipReadModeElementType> temperature_texture;
rtTextureSampler<float4,1,hipReadModeElementType> fire_color_texture;

__device__ float4 temperature_color_table(float normalized_temperature)
{
	return tex1D(fire_color_texture,normalized_temperature);
}

RT_PROGRAM void closest_hit_radiance_fire()
{
	PerRayData_radiance bg_prd;
	bg_prd.importance=1.0f;
	bg_prd.depth=0;
	optix::Ray bg_ray_radiance(ray.origin,ray.direction,RAY_TYPE_RADIANCE,RAY_T_MIN,RAY_T_MAX);
	rtTrace(top_opaque_object,bg_ray_radiance,bg_prd);
	float3 color=bg_prd.result;

	float3 start_point=ray.origin+back_box_hit_t*ray.direction;
	int n=int((back_box_hit_t-front_box_hit_t)/t_step);
	float absorption=4.0f*t_step;
	for(int i=0;i<n;i++){
		float3 pos=start_point-(float)i*t_step*ray.direction;
		float3 tex_coord=(pos-low_corner)*one_over_box_size;
		float sampled_density=tex3D(soot_texture,tex_coord.x,tex_coord.y,tex_coord.z);
		float normalized_sampled_temperature=tex3D(temperature_texture,tex_coord.x,tex_coord.y,tex_coord.z)/3000.0f;
		float attenuation=expf(-sampled_density*absorption);
		float3 emitting_color=make_float3(temperature_color_table(normalized_sampled_temperature));
		color=color*attenuation+0.3f*emitting_color*sampled_density;
	}

	prd.result=color;
}

RT_PROGRAM void closest_hit_radiance_smoke()
{
    float3 hit_point=ray.origin+front_box_hit_t*ray.direction;
    int n=int((back_box_hit_t-front_box_hit_t)/t_step);
    float color=0;
	float shadow=0;
    float alpha=0;
    float coef_density_to_alpha=0.8f;
	float coef_density_to_shadow=0.0f;
	float coef_density_to_color=1.0f;
	bool flag_compute_bg_color=true;
    for(int i=0;i<n;i++){
        float3 pos=hit_point+(float)i*t_step*ray.direction;
        float3 tex_coord=(pos-low_corner)*one_over_box_size;
        float sampled_density=tex3D(soot_texture,tex_coord.x,tex_coord.y,tex_coord.z);
		float sampled_alpha=1.0f-expf(-coef_density_to_alpha*sampled_density);
		float sampled_shadow=coef_density_to_shadow*sampled_density;
		float sampled_color=coef_density_to_color*sampled_density;	////not associtated with opacity yet
		alpha+=(1.0f-alpha)*sampled_alpha;
		if(alpha>1.0f){flag_compute_bg_color=false;break;}
		shadow=sampled_shadow+(1.0f-sampled_alpha)*shadow;
		if(shadow>1.0f){flag_compute_bg_color=false;break;}
		color+=(1.0f-alpha)*sampled_color*sampled_alpha*(1.0f-shadow);
    }

	float3 bg_color=make_float3(0,0,0);
	if(flag_compute_bg_color){
		PerRayData_radiance bg_prd;
		bg_prd.importance=1.0f;
		bg_prd.depth=0;
		optix::Ray bg_ray_radiance(ray.origin,ray.direction,RAY_TYPE_RADIANCE,RAY_T_MIN,RAY_T_MAX);
		rtTrace(top_opaque_object,bg_ray_radiance,bg_prd);
		bg_color=bg_prd.result;
	}

    prd.result=make_float3(color,color,color)+(1.0f-alpha)*bg_color;
}

RT_PROGRAM void closest_hit_shadow_smoke()
{
	float3 start_point=ray.origin+front_box_hit_t*ray.direction;
	int n=int((back_box_hit_t-front_box_hit_t)/t_step_2);
	float attenuation=1.0f;
	float coef_density_to_transpancy=0.2f;

	for(int i=0;i<n;i++){
		float3 pos=start_point+(float)i*t_step*ray.direction;
		float3 tex_coord=(pos-low_corner)*one_over_box_size;
		float sampled_density=tex3D(soot_texture,tex_coord.x,tex_coord.y,tex_coord.z);
		float sampled_transparency=expf(-coef_density_to_transpancy*sampled_density);	////more accurate
		attenuation*=sampled_transparency;
	}

	prd_shadow.attenuation*=attenuation;
}