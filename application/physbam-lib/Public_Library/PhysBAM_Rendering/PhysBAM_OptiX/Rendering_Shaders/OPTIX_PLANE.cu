#include "hip/hip_runtime.h"
//#####################################################################
// Copyright 2012, Bo Zhu
// This file is part of PhysBAM whose distribution is governed by the license contained in the accompanying file PHYSBAM_COPYRIGHT.txt.
//#####################################################################
#include <optix_world.h>
#include "OPTIX_HELPERS.h"
using namespace optix;

rtDeclareVariable(float,scene_epsilon,,);
rtDeclareVariable(optix::Ray,ray,rtCurrentRay,);
rtDeclareVariable(float3,geometric_normal,attribute geometric_normal,);
rtDeclareVariable(float3,shading_normal,attribute shading_normal,);

RT_PROGRAM void intersect(int)
{
    ////unlimited scale groud
    float tmin=(-scene_epsilon-ray.origin.y)/ray.direction.y;
    if(rtPotentialIntersection(tmin)){
        shading_normal=geometric_normal=make_float3(0,1.0f,0);
        rtReportIntersection(0);
    }
}

RT_PROGRAM void box_bounds(int,float result[6]) 
{
    optix::Aabb* aabb=(optix::Aabb*)result;
    aabb->invalidate(); ////infinite area
}

