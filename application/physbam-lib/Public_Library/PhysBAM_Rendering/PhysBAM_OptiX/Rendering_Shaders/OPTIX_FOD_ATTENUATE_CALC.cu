#include "hip/hip_runtime.h"
//#####################################################################
// Copyright 2011, Valeria Nikolaenko, Rahul Sheth
// This file is part of PhysBAM whose distribution is governed by the license contained in the accompanying file PHYSBAM_COPYRIGHT.txt.
//#####################################################################

#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>
#include "OPTIX_COMMONSTRUCTS.h"
#include "OPTIX_HELPERS.h"
#include "OPTIX_RAY_STRUCTS.h"
using namespace optix;

rtTextureSampler<float, 3, hipReadModeElementType> smoke_dencities_tex;
rtTextureSampler<float, 3, hipReadModeElementType> photon_map_tex;

rtDeclareVariable(float3,       low_corner, , );
rtDeclareVariable(float3,       up_corner, , );
rtDeclareVariable(float,       scene_epsilon, , );
rtDeclareVariable(float,       step, , );
rtDeclareVariable(float,       density_multiplier, , );
rtDeclareVariable(float,       exp_multiplier, , );
rtDeclareVariable(float,       exp_multiplier_2, , );
rtDeclareVariable(float,       exp_tex_resolution, , );

rtDeclareVariable(rtObject,          top_object, , );

rtDeclareVariable(rtObject,          top_opaque_object, , );

rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

rtDeclareVariable(float, front_box_hit_point, attribute front_box_hit_point, );
rtDeclareVariable(float, back_box_hit_point, attribute back_box_hit_point, );

__device__ bool isInsideDomain(float3 point) {
    return !(point.x < low_corner.x || point.x > up_corner.x ||
        point.y < low_corner.y || point.y > up_corner.y ||
        point.z < low_corner.z || point.z > up_corner.z);
}

__device__ float3 operator*(float3 a, uint3 b) {
    return make_float3(a.x * b.x, a.y * b.y, a.z * b.z);
}

__device__ float getIntensity(float3 point) {
    if (!isInsideDomain(point)) {
        return -1.f;
    }
    float3 tex_coord = (point - low_corner) / (up_corner - low_corner);
    return tex3D(smoke_dencities_tex, tex_coord.x, tex_coord.y, tex_coord.z);
}

RT_PROGRAM void any_hit_shadow() {
  // phongShadowed();
  rtIgnoreIntersection();
}

#if 0
RT_PROGRAM void closest_hit_radiance() {
  float3 hit_point = ray.origin + (t_hit + scene_epsilon) * ray.direction;

  PerRayData_radiance new_prd;
  new_prd.importance = prd.importance;
  new_prd.depth = prd.depth;
  optix::Ray next_ray = optix::make_Ray(hit_point, ray.direction, 0, scene_epsilon, RT_DEFAULT_MAX);
  rtTrace(top_opaque_object, next_ray, new_prd);

  float back_point = back_box_hit_point;//new_prd.distance > 0 ? (back_box_hit_point < new_prd.distance ? back_box_hit_point : new_prd.distance) : back_box_hit_point;
  /*
  if (new_prd.distance > 0) {
    back_box_hit_point = new_prd.distance;//back_box_hit_point < new_prd.distance ? back_box_hit_point : new_prd.distance;//fminf(back_box_hit_point, new_prd.distance);
  }
  */
  if (back_point - front_box_hit_point < scene_epsilon) {
      prd.result = new_prd.result;
      return;
  }

  float density = getIntensity(hit_point);
  float accumulative_density = 0;
  float accumulative_step = front_box_hit_point;
  if (density > 0) {
      do {
         accumulative_density += density * density_multiplier;
         accumulative_step += step;
         // hit_point += ray.direction * ;
         if (accumulative_density >= 1) {
            accumulative_density = 1;
            break;
         }
         density = getIntensity(hit_point + ray.direction * accumulative_step);
      } while (density >= 0.f && accumulative_step < front_box_hit_point + 2);

      // accumulative_density = back_point - front_box_hit_point;
      prd.result = make_float3(accumulative_density, accumulative_density, accumulative_density);// + new_prd.result;
  } else {
      prd.result = make_float3(1.f, 0.8f, 0.f);
  }
}

#endif

#if 0
RT_PROGRAM void closest_hit_radiance() {
  float3 hit_point = ray.origin + (t_hit + scene_epsilon) * ray.direction;

  if (!isInsideDomain(hit_point)) {
     prd.result = make_float3(1.f, 0.f, 0.f);
  }

  float density = getIntensity(hit_point);
  float accumulative_density;
  do {
     accumulative_density += density * density_multiplier;
     hit_point += ray.direction * step;
     if (accumulative_density >= 1) {
        accumulative_density = 1;
        break;
     }
     density = getIntensity(hit_point);
  } while (density >= 0.f);

  PerRayData_radiance new_prd;
  new_prd.importance = prd.importance * (1.f - accumulative_density);
  new_prd.depth = prd.depth;

  optix::Ray next_ray = optix::make_Ray(hit_point, ray.direction, 0, scene_epsilon, RT_DEFAULT_MAX);
  rtTrace(top_object, next_ray, new_prd);
  prd.result = make_float3(accumulative_density, accumulative_density, accumulative_density) + new_prd.result;
}
#endif

#if 0
RT_PROGRAM void closest_hit_radiance() {
  float3 hit_point = ray.origin + (t_hit + scene_epsilon) * ray.direction;


  float t = t_hit - front_box_hit_point;
  if (t > scene_epsilon || t < -scene_epsilon) {
    prd.result = make_float3(1.f, 0.f, 0.f);
    return;
  }

  PerRayData_radiance new_prd;
  new_prd.importance = prd.importance;
  new_prd.depth = prd.depth;

  optix::Ray next_ray = optix::make_Ray(ray.origin, ray.direction, 0, scene_epsilon, RT_DEFAULT_MAX);
  rtTrace(top_opaque_object, next_ray, new_prd);
  // prd.result = make_float3(accumulative_density, accumulative_density, accumulative_density) + new_prd.result;

  if (!isInsideDomain(hit_point)) {
     prd.result = new_prd.result;
     return;
  }

  float back_point = back_box_hit_point;
  if (new_prd.distance > 0) {
    back_point = fminf(back_box_hit_point, new_prd.distance);
  }

  float density = getIntensity(hit_point);
  float accumulative_density = 0;
  float accumulative_step = front_box_hit_point;
  do {
     accumulative_density += density * step;
     hit_point += ray.direction * step;
     if (accumulative_density >= 1) {
        accumulative_density = 1;
        break;
     }
     density = getIntensity(hit_point);
     accumulative_step += step;
  } while (density >= 0.f && accumulative_step < back_point);

  // PerRayData_radiance new_prd;
  /*new_prd.importance = prd.importance * (1.f - accumulative_density);
  new_prd.depth = prd.depth;

  optix::Ray next_ray = optix::make_Ray(hit_point, ray.direction, 0, scene_epsilon, RT_DEFAULT_MAX);
  rtTrace(top_object, next_ray, new_prd);
  */
  accumulative_density *= density_multiplier;
  prd.result = make_float3(accumulative_density, accumulative_density, accumulative_density) + new_prd.result;
}
#endif

RT_PROGRAM void closest_hit_radiance() {
  float3 hit_point = ray.origin + (t_hit + scene_epsilon) * ray.direction;

  float t = t_hit - front_box_hit_point;
  if (t > scene_epsilon || t < -scene_epsilon) {
    prd.result = make_float3(1.f, 0.f, 0.f);
    return;
  }

  PerRayData_radiance new_prd;
  new_prd.importance = prd.importance;
  new_prd.depth = prd.depth;

  optix::Ray next_ray = optix::make_Ray(ray.origin, ray.direction, 0, scene_epsilon, RT_DEFAULT_MAX);
  rtTrace(top_opaque_object, next_ray, new_prd);
  // prd.result = make_float3(accumulative_density, accumulative_density, accumulative_density) + new_prd.result;

  if (!isInsideDomain(hit_point)) {
     prd.result = new_prd.result;
     return;
  }

  float back_point = back_box_hit_point;
  if (new_prd.distance > 0) {
    back_point = fminf(back_box_hit_point, new_prd.distance);
  }

  float density = getIntensity(hit_point);
  float result_color_radiance = 0;
  float alpha = 1;

  // float accumulative_density = 0;
  float accumulative_step = front_box_hit_point;
  // float result_intensity = 0;
  float3 normalized_hit_point;

  do {
     normalized_hit_point = (hit_point - low_corner) / (up_corner - low_corner);
     result_color_radiance += alpha * density_multiplier * density * step * tex3D(photon_map_tex, normalized_hit_point.x, normalized_hit_point.y, normalized_hit_point.z);
     if (result_color_radiance > 1.0f) {
        result_color_radiance = 1.0f;
        break;
     }
     alpha *= (1 - density_multiplier * density * step);
     if (alpha < 0) {
        alpha = 0.f;
        break;
     }

     // result_intensity += tex3D(photon_map_tex, normalized_hit_point.x, normalized_hit_point.y, normalized_hit_point.z);// * tex1D(exp_tex, accumulative_density / exp_tex_resolution);

     // accumulative_density += density;
     hit_point += ray.direction * step;
     density = getIntensity(hit_point);
     accumulative_step += step;
  } while (density >= 0.f && accumulative_step < back_point);

  // result_intensity *= density_multiplier;
  // normalized_hit_point = (hit_point - low_corner) / (up_corner - low_corner);
  prd.result = make_float3(result_color_radiance, result_color_radiance, result_color_radiance) + alpha * new_prd.result;// + new_prd.result * exp_multiplier_2 * tex3D(photon_map_tex, normalized_hit_point.x, normalized_hit_point.y, normalized_hit_point.z) * tex1D(exp_tex, accumulative_density / exp_tex_resolution);
}
