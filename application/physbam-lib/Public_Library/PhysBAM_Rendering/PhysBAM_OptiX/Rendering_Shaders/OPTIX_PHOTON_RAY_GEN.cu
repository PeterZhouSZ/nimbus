#include "hip/hip_runtime.h"
//#####################################################################
// Copyright 2011, Valeria Nikolaenko, Rahul Sheth
// This file is part of PhysBAM whose distribution is governed by the license contained in the accompanying file PHYSBAM_COPYRIGHT.txt.
//#####################################################################

#include <optix_world.h>
#include "OPTIX_COMMONSTRUCTS.h"
#include "OPTIX_RAY_STRUCTS.h"
#include "OPTIX_HELPERS.h"
using namespace optix;

rtTextureSampler<float, 3, hipReadModeElementType> smoke_dencities_tex2;
rtBuffer<float, 3> photon_map;

rtBuffer<BasicLight>                 lights;
rtDeclareVariable(float3,       low_corner, , );
rtDeclareVariable(float3,       up_corner, , );
rtDeclareVariable(float,       scene_epsilon, , );
rtDeclareVariable(float3,       absorption, , );
rtDeclareVariable(float3,       scattering, , );
rtDeclareVariable(float,       step_size, , );
rtDeclareVariable(rtObject, smoke_object, , );
rtDeclareVariable(float,       step, , );
rtDeclareVariable(float,       color_multiplier, , );

rtDeclareVariable(uint3, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint3, launch_dim, rtLaunchDim, );
rtDeclareVariable(rtObject,          top_opaque_object, , );

__device__ bool isInsideDomain(float3 point) {
    return !(point.x < low_corner.x || point.x > up_corner.x ||
        point.y < low_corner.y || point.y > up_corner.y ||
        point.z < low_corner.z || point.z > up_corner.z);
}

__device__ float3 operator*(float3 a, uint3 b) {
    return make_float3(a.x * b.x, a.y * b.y, a.z * b.z);
}

__device__ float getIntensity(float3 point) {
    if (!isInsideDomain(point)) {
        return -1.f;
    }
    float3 tex_coord = (point - low_corner) / (up_corner - low_corner);
    return tex3D(smoke_dencities_tex2, tex_coord.x, tex_coord.y, tex_coord.z);
}

__device__ float intersect(optix::Ray ray) {
    float3 t1 = (up_corner - ray.origin) / ray.direction;
    float3 t2 = (low_corner - ray.origin) / ray.direction;

    float3 T1 = fminf(t1, t2);
    float3 T2 = fmaxf(t1, t2);

    float Tnear = fmaxf(T1);
    float Tfar = fminf(T2);

    if (Tnear < Tfar && Tfar > 0) {
        if (Tnear < 0) {
            Tnear = scene_epsilon * 10;
        }
        return Tnear;
    }
    return -1;
}

__device__ float attenuation(optix::Ray ray, float t_hit, float stop_distance) {
  float3 hit_point = ray.origin + (t_hit + scene_epsilon) * ray.direction;

  PerRayData_radiance new_prd;
  new_prd.depth = 0;
  new_prd.importance = 1.f;

  rtTrace(top_opaque_object, ray, new_prd);

  if (!isInsideDomain(hit_point)) {
     return 0.f;
  }

  float alpha = 1.0f;
  if (new_prd.distance > 0 && new_prd.distance < stop_distance) {
    alpha = 0.5f;
  }

  float density = getIntensity(hit_point);
  float accumulative_density = 0;
  float accumulative_step = t_hit;
  do {
     accumulative_density += density * step;
     hit_point += ray.direction * step;
     density = getIntensity(hit_point);
     accumulative_step += step;
  } while (density >= 0.f && accumulative_step < stop_distance);

  float res = color_multiplier * accumulative_density;//exp(-density_multiplier * accumulative_density);
  if (res > 1.0f) {
    res = 1.0f;
  } else if (res < 0.f) {
    res = 0.f;
  }
  return (1.0f - res) * alpha;
}

RT_PROGRAM void ray_gen() {
  unsigned int num_lights = lights.size();
  float3 offsets[4] = {make_float3(0.25f, 0.25f, 0.25f),
                       // make_float3(-0.25f, 0.25f, 0.25f),
                       make_float3(-0.25f, -0.25f, 0.25f),
                       // make_float3(0.25f, -0.25f, 0.25f),

                       // make_float3(0.25f, 0.25f, -0.25f),
                       make_float3(-0.25f, 0.25f, -0.25f),
                       // make_float3(-0.25f, -0.25f, -0.25f),
                       make_float3(0.25f, -0.25f, -0.25f)};

  photon_map[launch_index] = 0.f;
  for(int i = 0; i < num_lights; ++i) {
      // photon_map[launch_index] += 1.0f;

      for (int j = 0; j < 4; j++) {
          float3 grid_point = ((make_float3(launch_index) + make_float3(0.5f, 0.5f, 0.5f) + offsets[j]) / make_float3(launch_dim)) * (up_corner - low_corner) + low_corner;
          optix::Ray ray = optix::make_Ray(lights[i].pos, normalize(grid_point - lights[i].pos), 0, scene_epsilon, RT_DEFAULT_MAX);

          // rtTrace(smoke_object, ray, prd);
          float distance = intersect(ray);
          if (distance > 0) {
            photon_map[launch_index] += 0.25f * attenuation(ray, distance, length(lights[i].pos - grid_point));
          }
      }
      // accumulating photons from all light sources
      // prd.result = 1.0f;
      // photon_map[launch_index] += prd.result;
  }
}

RT_PROGRAM void exception() {
  const unsigned int code = rtGetExceptionCode();
  rtPrintf( "Caught exception during photon mapping 0x%X at launch index (%d,%d,%d)\n", code, launch_index.x, launch_index.y, launch_index.z);
  photon_map[launch_index] = 0.f;
}


