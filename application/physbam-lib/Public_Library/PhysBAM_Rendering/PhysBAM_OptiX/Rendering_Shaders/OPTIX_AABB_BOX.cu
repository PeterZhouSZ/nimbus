#include "hip/hip_runtime.h"
//#####################################################################
// Copyright 2011, Valeria Nikolaenko, Rahul Sheth
// This file is part of PhysBAM whose distribution is governed by the license contained in the accompanying file PHYSBAM_COPYRIGHT.txt.
//#####################################################################
#include <optix_world.h>
#include "OPTIX_HELPERS.h"
#include "OPTIX_RAY_STRUCTS.h"
using namespace optix;

rtDeclareVariable(float3,low_corner,,);
rtDeclareVariable(float3,up_corner,,);
rtDeclareVariable(float,scene_epsilon,,);
rtDeclareVariable(optix::Ray,ray,rtCurrentRay,);
rtDeclareVariable(float,front_box_hit_t,attribute front_box_hit_t,);
rtDeclareVariable(float,back_box_hit_t,attribute back_box_hit_t,);
rtDeclareVariable(float3,back_hit_point,attribute back_hit_point,);
rtDeclareVariable(float3,front_hit_point,attribute front_hit_point,);

__device__ bool operator<(float3 a,float3 b){return a.x<b.x&&a.y<b.y&&a.z<b.z;}
__device__ bool operator>(float3 a,float3 b){return a.x > b.x&&a.y>b.y&&a.z>b.z;}

RT_PROGRAM void intersect(int) 
{
    float3 t1=(up_corner-ray.origin)/ray.direction;
    float3 t2=(low_corner-ray.origin)/ray.direction;
    float3 T1=fminf(t1,t2);
    float3 T2=fmaxf(t1,t2);
    float Tnear=fmaxf(T1);
    float Tfar=fminf(T2);

    if (Tnear<Tfar&&Tfar>0){
        if (Tnear<0){Tnear=RAY_T_MIN*2;}
        if(rtPotentialIntersection(Tnear)){
            back_hit_point=front_hit_point=ray.origin+Tnear*ray.direction;
            front_box_hit_t=Tnear;
            back_box_hit_t=Tfar;
            rtReportIntersection(0);
        }
    }
}

RT_PROGRAM void box_bounds(int,float result[6]) 
{
  optix::Aabb* aabb=(optix::Aabb*)result;
  aabb->m_min=low_corner;
  aabb->m_max=up_corner;
}

